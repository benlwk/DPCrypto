#include "hip/hip_runtime.h"

#include "../include/poly.cuh"
#include "../include/cuda_kernel.cuh"
#include <stdlib.h> 

#define SHAKE128_RATE 168
#define SHAKE256_RATE 136
#define SHA3_256_RATE 136
#define SHA3_512_RATE 72
#define MODN(X) ((X) & (SABER_N-1))   
#define h1 (1 << (SABER_EQ - SABER_EP - 1))
#define h2 ((1 << (SABER_EP - 2)) - (1 << (SABER_EP - SABER_ET - 1)) + (1 << (SABER_EQ - SABER_EP - 1)))

__global__ void packdp2av1(short2 *outa, uint16_t *a) 
{
    int i;
    uint32_t tid = threadIdx.x, bid = blockIdx.x;

    for(i=0; i<SABER_N/2; i++)
    {
      outa[bid * SABER_N/2 + i*SABER_N/2 + tid].x = a[bid * SABER_N + MODN(tid - i*2)]; 
      outa[bid * SABER_N/2 + i*SABER_N/2 + tid].y = a[bid * SABER_N + MODN(tid - i*2 - 1)];
    }    
}

__global__ void packdp2av2(short2 *outa1, short2 *outa2, uint16_t *a) 
{    
    int32_t tid = threadIdx.x, bid = blockIdx.x*SABER_L*SABER_L*SABER_N/2, bid2 = blockIdx.x*SABER_L*SABER_L*SABER_N;

   int i, j;
   
   for (i = 0; i < SABER_L; i++)
   {
      for (j = 0; j < SABER_L; j++)
      {     
         outa1[bid  + i*SABER_N*SABER_L/2 +j*SABER_N/2 + tid].x = a[bid2+ i*SABER_N*SABER_L+j*SABER_N + tid*2 + 1];  
         outa1[bid  + i*SABER_N*SABER_L/2 +j*SABER_N/2+ tid].y = a[bid2+ i*SABER_N*SABER_L+j*SABER_N + tid*2]; 
         outa2[bid  + i*SABER_N*SABER_L/2 +j*SABER_N/2+ tid].x = a[bid2+ i*SABER_N*SABER_L+j*SABER_N + (SABER_N - tid*2)%SABER_N];      
         outa2[bid  + i*SABER_N*SABER_L/2 +j*SABER_N/2+ tid].y = a[bid2+ i*SABER_N*SABER_L+j*SABER_N + (SABER_N - tid*2 - 1)%SABER_N ];           
      }
   }
}

__global__ void packdp2av2Inner(short2 *outa1, short2 *outa2, uint16_t *a) 
{    
    int32_t tid = threadIdx.x, bid = blockIdx.x*SABER_L*SABER_N/2, bid2 = blockIdx.x*SABER_L*SABER_N;

   int j;
   
     for (j = 0; j < SABER_L; j++)
      {     
         outa1[bid  + j*SABER_N/2 + tid].x = a[bid2+ j*SABER_N + tid*2 + 1];  
         outa1[bid  + j*SABER_N/2+ tid].y = a[bid2+ j*SABER_N + tid*2]; 
         outa2[bid  + j*SABER_N/2+ tid].x = a[bid2+ j*SABER_N + (SABER_N - tid*2)%SABER_N];      
         outa2[bid  + j*SABER_N/2+ tid].y = a[bid2+ j*SABER_N + (SABER_N - tid*2 - 1)%SABER_N ];
      }
}
__global__ void packdp2b(char4 *outb, uint16_t* b) 
{
   uint32_t tid = threadIdx.x, bid = blockIdx.x*SABER_L*SABER_N/2, bid2 = blockIdx.x*SABER_L*SABER_N;
   int j;
   
   for (j = 0; j < SABER_L; j++)
   { 
      outb[bid  +j*SABER_N/2 + tid].x = b[bid2+j*SABER_N + tid*2]; 
      outb[bid  +j*SABER_N/2 + tid].y = b[bid2+j*SABER_N + tid*2+1];    
   }  
}

// DoDP4Av4 with no C intrinsic, handle odd
__global__ void DoDP4Av7(uint16_t *out, short2 *a1, short2 *a2, char4* b) {
    uint32_t tid = threadIdx.x, bid = blockIdx.x*SABER_L*SABER_L*SABER_N/2, bid2 = blockIdx.x*SABER_L*SABER_N, bid3 = blockIdx.x*SABER_L*SABER_N/2;
    int i, j, k;
    int32_t sum1 = 0, sum2 = 0;
    ushort2 load_a;
    __shared__ ushort2 s_a1[SABER_N], s_a2[SABER_N];
    __shared__ char4 s_b[SABER_N];

   for (j = 0; j < SABER_L; j++)
   { 
       s_b[tid].x = b[bid3 +j*SABER_N/2 + tid].x;   
       s_b[tid].y = b[bid3 +j*SABER_N/2 + tid].y;
       // s_b[tid + SABER_N/2].x = b[bid3 +j*SABER_N/2 + tid + (SABER_N)/2].x;
       // s_b[tid + SABER_N/2].y = b[bid3 +j*SABER_N/2 + tid + (SABER_N)/2].y;      
      for (i = 0; i < SABER_L; i++)
      {
    
       s_a1[tid].x = a1[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid].x;
       s_a1[tid + (SABER_N)/2].x = a1[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid + (SABER_N)/2].x;
       s_a2[tid].x = a2[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid].x;
       s_a2[tid + (SABER_N)/2].x = a2[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid + (SABER_N)/2].x;

       s_a1[tid].y = a1[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid].y;
       s_a1[tid + (SABER_N)/2].y = a1[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid + (SABER_N)/2].y;
       s_a2[tid].y = a2[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid].y;
       s_a2[tid + (SABER_N)/2].y = a2[bid + i*SABER_N*SABER_L/2+j*SABER_N/2 + tid + (SABER_N)/2].y;
       __syncthreads();

       sum1 = 0;
       sum2 = 0;
       for(k=0; k<SABER_N/2; k++)
       { 
            // Processing even element
            load_a = s_a2[(tid*(SABER_N/2-1) + k)%(SABER_N/2)]; 
           if(k > tid)
           {               
               sum1 -= load_a.x*s_b[k].x + load_a.y * s_b[k].y;
               // if(tid==0) printf("-tid: %u %u += %u * %u + %u * %u\n", tid, sum1, load_a.x, s_b[k].x, load_a.y, s_b[k].y);     
           }
           else if (k==tid)
           {
               sum1 += load_a.x * s_b[k].x; 
               sum1 -= load_a.y * s_b[k].y; 
               // if(tid==0) printf("+*tid: %u %u += %u * %u - %u * %u\n", tid, sum1, load_a.x, s_b[k].x, load_a.y, s_b[k].y);    
           }
           else
           {
               sum1 += load_a.x*s_b[k].x + load_a.y * s_b[k].y; 
               // if(tid==0) printf("+tid: %u %u += %u * %u + %u * %u\n", tid, sum1, load_a.x, s_b[k].x, load_a.y, s_b[k].y);           
           }
           // Processing odd element
           load_a = s_a1[(tid + k*(SABER_N/2-1))%(SABER_N/2)];
           if(k<=tid)
           {               
               sum2 += load_a.x*s_b[k].x + load_a.y * s_b[k].y;  
               // if(tid==1) printf("+tid: %u %u += %u * %u + %u * %u\n", tid, sum2, load_a.x, s_b[k].x, load_a.y, s_b[k].y);  
           }
           else
           {
               sum2 -= load_a.x*s_b[k].x + load_a.y * s_b[k].y;  
               // if(tid==1) printf("-tid: %u %u += %u * %u + %u * %u\n", tid, sum2, load_a.x, s_b[k].x, load_a.y, s_b[k].y);  
           }
       }
       __syncthreads();
       // out[bid2 + i*SABER_N +tid*2] += MODQ(sum1);   
       // out[bid2 + i*SABER_N +tid*2+1] += MODQ(sum2);  
       out[bid2 + i*SABER_N +tid*2] += sum1;   
       out[bid2 + i*SABER_N +tid*2+1] += sum2;        
      }
   } 
}


__global__ void DoDP4Av7_inner(uint16_t *out, short2 *a1, short2 *a2, char4* b) {
    uint32_t tid = threadIdx.x, bid = blockIdx.x*SABER_L*SABER_N/2, bid2 = blockIdx.x*SABER_N, bid3 = blockIdx.x*SABER_L*SABER_N/2;
    int i, k;
    int32_t sum1 = 0, sum2 = 0;
    ushort2 load_a;
    __shared__ ushort2 s_a1[SABER_N], s_a2[SABER_N];
    __shared__ char4 s_b[SABER_N];

   for (i = 0; i < SABER_L; i++)
   {
      s_b[tid].x = b[bid3 +i*SABER_N/2 + tid].x;   
      s_b[tid].y = b[bid3 +i*SABER_N/2 + tid].y;
      s_a1[tid].x = a1[bid + i*SABER_N/2 + tid].x;
      s_a1[tid + SABER_N/2].x = a1[bid + i*SABER_N/2 + tid + (SABER_N)/2].x;
      s_a2[tid].x = a2[bid + i*SABER_N/2 + tid].x;
      s_a2[tid + SABER_N/2].x = a2[bid + i*SABER_N/2 + tid + (SABER_N)/2].x;

      s_a1[tid].y = a1[bid + i*SABER_N/2 + tid].y;
      s_a1[tid + SABER_N/2].y = a1[bid + i*SABER_N/2 + tid + SABER_N/2].y;
      s_a2[tid].y = a2[bid + i*SABER_N/2 + tid].y;
      s_a2[tid + SABER_N/2].y = a2[bid + i*SABER_N/2 + tid + SABER_N/2].y;
      __syncthreads();

      sum1 = 0;
      sum2 = 0;
      for(k=0; k<SABER_N/2; k++)
      { 
           if(k > tid)
           {
               load_a = s_a2[(tid*(SABER_N/2-1) + k)%(SABER_N/2)]; 
               sum1 -= load_a.x*s_b[k].x + load_a.y * s_b[k].y;    
           }
           else if (k==tid)
           {
               load_a = s_a2[(tid*(SABER_N/2-1) + k)%(SABER_N/2)];  
               sum1 += load_a.x * s_b[k].x; 
               sum1 -= load_a.y * s_b[k].y; 
           }
           else
           {
               load_a = s_a2[(tid*(SABER_N/2-1) + k)%(SABER_N/2)];  
               sum1 += load_a.x*s_b[k].x + load_a.y * s_b[k].y;    
           }
           
           if(k<=tid)
           {
               load_a = s_a1[(tid + k*(SABER_N/2-1))%(SABER_N/2)];
               sum2 += load_a.x*s_b[k].x + load_a.y * s_b[k].y;
           }
           else
           {
               load_a = s_a1[(tid + k*(SABER_N/2-1))%(SABER_N/2)];
               sum2 -= load_a.x*s_b[k].x + load_a.y * s_b[k].y; 
           }
       }
       __syncthreads();
       // out[bid2 + tid*2] += MODQ(sum1);   
       // out[bid2 + tid*2+1] += MODQ(sum2);  
       out[bid2 + tid*2] += sum1;   
       out[bid2 + tid*2+1] += sum2;        
      
   } 
}


__global__ void VecVecMul_gpu_shared(uint16_t *r, uint16_t *g_a, uint16_t *g_s)
{
   int16_t j, k, sum;
   uint32_t tid = threadIdx.x, bidx1 = blockIdx.x * SABER_N*SABER_L;
   uint32_t bidx2 = blockIdx.x * SABER_N;
   __shared__ int16_t s0[SABER_L*SABER_N], a[SABER_N];
   for (j = 0; j < SABER_L; j++)
   {
      a[tid] = g_a[bidx1  + j*SABER_N + tid];
      s0[tid] = g_s[bidx1 + j*SABER_N + tid];   
      __syncthreads();
      sum = 0;// use register to accumulate
      for(k=0; k<tid+1; k++)
         sum += s0[tid-k] * a[k];  
      for(k=1; k<SABER_N-tid; k++)
         sum -= s0[tid+k] * a[(SABER_N)-k];   
       __syncthreads();
      // r[bidx2 + tid] +=MODQ(sum) ;       
       r[bidx2 + tid] +=sum ;     
   }
  
 }
__global__ void MatVecMul_gpu_shared2(uint16_t *r, uint16_t *g_a, uint16_t *g_s)
{
   int16_t k, sum;
   uint32_t tid = threadIdx.x, bidx1 = blockIdx.x * SABER_N*SABER_L*SABER_L;
   uint32_t bidx2 = blockIdx.x * SABER_N*SABER_L;
   __shared__ int16_t s0[SABER_N], a[SABER_N];
   
   a[tid] = g_a[bidx1 + tid];
   s0[tid] = g_s[bidx2 + tid];   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s0[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s0[tid+k] * a[(SABER_N)-k];   
    __syncthreads();
   // r[bidx2 + tid] +=MODQ(sum) ;       
    r[bidx2 + tid] +=sum ;       
 }

// Only for Saber, not for FireSaber or LightSaber
__global__ void MatVecMul_gpu_shared(uint16_t *r, uint16_t *g_a, uint16_t *g_s)
{
   uint16_t k, sum;
   uint32_t tid = threadIdx.x, bidx1 = blockIdx.x * SABER_N*SABER_L*SABER_L;
   uint32_t bidx2 = blockIdx.x * SABER_N*SABER_L;
   __shared__ uint16_t s0[SABER_N], s1[SABER_N], s2[SABER_N], a[SABER_N];

   // i=0, j=0
   a[tid] = g_a[bidx1 + tid];
   s0[tid] = g_s[bidx2 + tid];   
   __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s0[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s0[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + tid] +=sum ;    
   __syncthreads();

    // i=0, j=1
   a[tid] = g_a[bidx1 + SABER_N + tid];
   s1[tid] = g_s[bidx2 + SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s1[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s1[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + tid] +=sum ;
   __syncthreads();

    // i=0, j=2
   a[tid] = g_a[bidx1 + 2*SABER_N + tid];
   s2[tid] = g_s[bidx2 + 2*SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s2[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s2[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + tid] +=sum ;
   __syncthreads();

    // i=1, j=0
   a[tid] = g_a[bidx1 + SABER_L*SABER_N + tid];   
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s0[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s0[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + SABER_N + tid] +=sum ;
   __syncthreads();

    // i=1, j=1
   a[tid] = g_a[bidx1 + SABER_L*SABER_N + SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s1[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s1[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + SABER_N + tid] +=sum ;
   __syncthreads();

    // i=1, j=2
   a[tid] = g_a[bidx1 + SABER_L*SABER_N + 2*SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s2[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s2[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + SABER_N + tid] +=sum ;
   __syncthreads();

    // i=2, j=0
   a[tid] = g_a[bidx1 + 2*SABER_L*SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s0[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s0[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + 2*SABER_N + tid] +=sum ;
   __syncthreads();

    // i=2, j=1
   a[tid] = g_a[bidx1 + 2*SABER_L*SABER_N + SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s1[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s1[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + 2*SABER_N + tid] +=sum ;
   __syncthreads();

    // i=2, j=2
   a[tid] = g_a[bidx1 + 2*SABER_L*SABER_N + 2*SABER_N + tid];
    __syncthreads();   
   sum = 0;// use register to accumulate
   for(k=0; k<tid+1; k++)
      sum += s2[tid-k] * a[k];  
   for(k=1; k<SABER_N-tid; k++)
      sum -= s2[tid+k] * a[(SABER_N)-k];   
   r[bidx2 + 2*SABER_N + tid] +=sum ;
}

__global__ void post_process(uint16_t *in)
{
   uint32_t tid = threadIdx.x, bid = blockIdx.x * SABER_N*SABER_L; 
   int i;
   for (i = 0; i < SABER_L; i++)
   {
      in[bid + i*SABER_N + tid] = (in[bid +i*SABER_N + tid] + h1) >> (SABER_EQ - SABER_EP);
   }
}

__global__ void post_process2(uint16_t *out, uint16_t *in)
{
   uint32_t tid = threadIdx.x, bid = blockIdx.x*SABER_N; 
   out[bid + tid] = (out[bid + tid] - (in[bid + tid] << (SABER_EP - 1)) + h1) >> (SABER_EP - SABER_ET);
}


__global__ void post_process3(uint16_t *out, uint16_t *in)
{
   uint32_t tid = threadIdx.x, bid = blockIdx.x*SABER_N; 

   out[bid + tid] = (out[bid + tid] + h2 - (in[bid + tid] << (SABER_EP - SABER_ET))) >> (SABER_EP - 1);
}

#define NROUNDS 24
#define ROL(a, offset) ((a << offset) ^ (a >> (64 - offset)))

__device__ static uint64_t load64(const unsigned char *x)
{
  unsigned long long r = 0, i;

  for (i = 0; i < 8; ++i)
  {
    r |= (unsigned long long)x[i] << 8 * i;
  }
  return r;
}

__device__ static void store64(uint8_t *x, uint64_t u)
{
  unsigned int i;

  for (i = 0; i < 8; ++i)
  {
    x[i] = u;
    u >>= 8;
  }
}

__constant__ const uint64_t KeccakF_RoundConstants[NROUNDS] =
    {
        (uint64_t)0x0000000000000001ULL,
        (uint64_t)0x0000000000008082ULL,
        (uint64_t)0x800000000000808aULL,
        (uint64_t)0x8000000080008000ULL,
        (uint64_t)0x000000000000808bULL,
        (uint64_t)0x0000000080000001ULL,
        (uint64_t)0x8000000080008081ULL,
        (uint64_t)0x8000000000008009ULL,
        (uint64_t)0x000000000000008aULL,
        (uint64_t)0x0000000000000088ULL,
        (uint64_t)0x0000000080008009ULL,
        (uint64_t)0x000000008000000aULL,
        (uint64_t)0x000000008000808bULL,
        (uint64_t)0x800000000000008bULL,
        (uint64_t)0x8000000000008089ULL,
        (uint64_t)0x8000000000008003ULL,
        (uint64_t)0x8000000000008002ULL,
        (uint64_t)0x8000000000000080ULL,
        (uint64_t)0x000000000000800aULL,
        (uint64_t)0x800000008000000aULL,
        (uint64_t)0x8000000080008081ULL,
        (uint64_t)0x8000000000008080ULL,
        (uint64_t)0x0000000080000001ULL,
        (uint64_t)0x8000000080008008ULL};

__device__ static void KeccakF1600_StatePermute(uint64_t *state)
{
  int round;

  uint64_t Aba, Abe, Abi, Abo, Abu;
  uint64_t Aga, Age, Agi, Ago, Agu;
  uint64_t Aka, Ake, Aki, Ako, Aku;
  uint64_t Ama, Ame, Ami, Amo, Amu;
  uint64_t Asa, Ase, Asi, Aso, Asu;
  uint64_t BCa, BCe, BCi, BCo, BCu;
  uint64_t Da, De, Di, Do, Du;
  uint64_t Eba, Ebe, Ebi, Ebo, Ebu;
  uint64_t Ega, Ege, Egi, Ego, Egu;
  uint64_t Eka, Eke, Eki, Eko, Eku;
  uint64_t Ema, Eme, Emi, Emo, Emu;
  uint64_t Esa, Ese, Esi, Eso, Esu;

  //copyFromState(A, state)
  Aba = state[0];
  Abe = state[1];
  Abi = state[2];
  Abo = state[3];
  Abu = state[4];
  Aga = state[5];
  Age = state[6];
  Agi = state[7];
  Ago = state[8];
  Agu = state[9];
  Aka = state[10];
  Ake = state[11];
  Aki = state[12];
  Ako = state[13];
  Aku = state[14];
  Ama = state[15];
  Ame = state[16];
  Ami = state[17];
  Amo = state[18];
  Amu = state[19];
  Asa = state[20];
  Ase = state[21];
  Asi = state[22];
  Aso = state[23];
  Asu = state[24];

  for (round = 0; round < NROUNDS; round += 2)
  {
    //    prepareTheta
    BCa = Aba ^ Aga ^ Aka ^ Ama ^ Asa;
    BCe = Abe ^ Age ^ Ake ^ Ame ^ Ase;
    BCi = Abi ^ Agi ^ Aki ^ Ami ^ Asi;
    BCo = Abo ^ Ago ^ Ako ^ Amo ^ Aso;
    BCu = Abu ^ Agu ^ Aku ^ Amu ^ Asu;

    //thetaRhoPiChiIotaPrepareTheta(round  , A, E)
    Da = BCu ^ ROL(BCe, 1);
    De = BCa ^ ROL(BCi, 1);
    Di = BCe ^ ROL(BCo, 1);
    Do = BCi ^ ROL(BCu, 1);
    Du = BCo ^ ROL(BCa, 1);

    Aba ^= Da;
    BCa = Aba;
    Age ^= De;
    BCe = ROL(Age, 44);
    Aki ^= Di;
    BCi = ROL(Aki, 43);
    Amo ^= Do;
    BCo = ROL(Amo, 21);
    Asu ^= Du;
    BCu = ROL(Asu, 14);
    Eba = BCa ^ ((~BCe) & BCi);
    Eba ^= (uint64_t)KeccakF_RoundConstants[round];
    Ebe = BCe ^ ((~BCi) & BCo);
    Ebi = BCi ^ ((~BCo) & BCu);
    Ebo = BCo ^ ((~BCu) & BCa);
    Ebu = BCu ^ ((~BCa) & BCe);

    Abo ^= Do;
    BCa = ROL(Abo, 28);
    Agu ^= Du;
    BCe = ROL(Agu, 20);
    Aka ^= Da;
    BCi = ROL(Aka, 3);
    Ame ^= De;
    BCo = ROL(Ame, 45);
    Asi ^= Di;
    BCu = ROL(Asi, 61);
    Ega = BCa ^ ((~BCe) & BCi);
    Ege = BCe ^ ((~BCi) & BCo);
    Egi = BCi ^ ((~BCo) & BCu);
    Ego = BCo ^ ((~BCu) & BCa);
    Egu = BCu ^ ((~BCa) & BCe);

    Abe ^= De;
    BCa = ROL(Abe, 1);
    Agi ^= Di;
    BCe = ROL(Agi, 6);
    Ako ^= Do;
    BCi = ROL(Ako, 25);
    Amu ^= Du;
    BCo = ROL(Amu, 8);
    Asa ^= Da;
    BCu = ROL(Asa, 18);
    Eka = BCa ^ ((~BCe) & BCi);
    Eke = BCe ^ ((~BCi) & BCo);
    Eki = BCi ^ ((~BCo) & BCu);
    Eko = BCo ^ ((~BCu) & BCa);
    Eku = BCu ^ ((~BCa) & BCe);

    Abu ^= Du;
    BCa = ROL(Abu, 27);
    Aga ^= Da;
    BCe = ROL(Aga, 36);
    Ake ^= De;
    BCi = ROL(Ake, 10);
    Ami ^= Di;
    BCo = ROL(Ami, 15);
    Aso ^= Do;
    BCu = ROL(Aso, 56);
    Ema = BCa ^ ((~BCe) & BCi);
    Eme = BCe ^ ((~BCi) & BCo);
    Emi = BCi ^ ((~BCo) & BCu);
    Emo = BCo ^ ((~BCu) & BCa);
    Emu = BCu ^ ((~BCa) & BCe);

    Abi ^= Di;
    BCa = ROL(Abi, 62);
    Ago ^= Do;
    BCe = ROL(Ago, 55);
    Aku ^= Du;
    BCi = ROL(Aku, 39);
    Ama ^= Da;
    BCo = ROL(Ama, 41);
    Ase ^= De;
    BCu = ROL(Ase, 2);
    Esa = BCa ^ ((~BCe) & BCi);
    Ese = BCe ^ ((~BCi) & BCo);
    Esi = BCi ^ ((~BCo) & BCu);
    Eso = BCo ^ ((~BCu) & BCa);
    Esu = BCu ^ ((~BCa) & BCe);

    //    prepareTheta
    BCa = Eba ^ Ega ^ Eka ^ Ema ^ Esa;
    BCe = Ebe ^ Ege ^ Eke ^ Eme ^ Ese;
    BCi = Ebi ^ Egi ^ Eki ^ Emi ^ Esi;
    BCo = Ebo ^ Ego ^ Eko ^ Emo ^ Eso;
    BCu = Ebu ^ Egu ^ Eku ^ Emu ^ Esu;

    //thetaRhoPiChiIotaPrepareTheta(round+1, E, A)
    Da = BCu ^ ROL(BCe, 1);
    De = BCa ^ ROL(BCi, 1);
    Di = BCe ^ ROL(BCo, 1);
    Do = BCi ^ ROL(BCu, 1);
    Du = BCo ^ ROL(BCa, 1);

    Eba ^= Da;
    BCa = Eba;
    Ege ^= De;
    BCe = ROL(Ege, 44);
    Eki ^= Di;
    BCi = ROL(Eki, 43);
    Emo ^= Do;
    BCo = ROL(Emo, 21);
    Esu ^= Du;
    BCu = ROL(Esu, 14);
    Aba = BCa ^ ((~BCe) & BCi);
    Aba ^= (uint64_t)KeccakF_RoundConstants[round + 1];
    Abe = BCe ^ ((~BCi) & BCo);
    Abi = BCi ^ ((~BCo) & BCu);
    Abo = BCo ^ ((~BCu) & BCa);
    Abu = BCu ^ ((~BCa) & BCe);

    Ebo ^= Do;
    BCa = ROL(Ebo, 28);
    Egu ^= Du;
    BCe = ROL(Egu, 20);
    Eka ^= Da;
    BCi = ROL(Eka, 3);
    Eme ^= De;
    BCo = ROL(Eme, 45);
    Esi ^= Di;
    BCu = ROL(Esi, 61);
    Aga = BCa ^ ((~BCe) & BCi);
    Age = BCe ^ ((~BCi) & BCo);
    Agi = BCi ^ ((~BCo) & BCu);
    Ago = BCo ^ ((~BCu) & BCa);
    Agu = BCu ^ ((~BCa) & BCe);

    Ebe ^= De;
    BCa = ROL(Ebe, 1);
    Egi ^= Di;
    BCe = ROL(Egi, 6);
    Eko ^= Do;
    BCi = ROL(Eko, 25);
    Emu ^= Du;
    BCo = ROL(Emu, 8);
    Esa ^= Da;
    BCu = ROL(Esa, 18);
    Aka = BCa ^ ((~BCe) & BCi);
    Ake = BCe ^ ((~BCi) & BCo);
    Aki = BCi ^ ((~BCo) & BCu);
    Ako = BCo ^ ((~BCu) & BCa);
    Aku = BCu ^ ((~BCa) & BCe);

    Ebu ^= Du;
    BCa = ROL(Ebu, 27);
    Ega ^= Da;
    BCe = ROL(Ega, 36);
    Eke ^= De;
    BCi = ROL(Eke, 10);
    Emi ^= Di;
    BCo = ROL(Emi, 15);
    Eso ^= Do;
    BCu = ROL(Eso, 56);
    Ama = BCa ^ ((~BCe) & BCi);
    Ame = BCe ^ ((~BCi) & BCo);
    Ami = BCi ^ ((~BCo) & BCu);
    Amo = BCo ^ ((~BCu) & BCa);
    Amu = BCu ^ ((~BCa) & BCe);

    Ebi ^= Di;
    BCa = ROL(Ebi, 62);
    Ego ^= Do;
    BCe = ROL(Ego, 55);
    Eku ^= Du;
    BCi = ROL(Eku, 39);
    Ema ^= Da;
    BCo = ROL(Ema, 41);
    Ese ^= De;
    BCu = ROL(Ese, 2);
    Asa = BCa ^ ((~BCe) & BCi);
    Ase = BCe ^ ((~BCi) & BCo);
    Asi = BCi ^ ((~BCo) & BCu);
    Aso = BCo ^ ((~BCu) & BCa);
    Asu = BCu ^ ((~BCa) & BCe);
  }

  //copyToState(state, A)
  state[0] = Aba;
  state[1] = Abe;
  state[2] = Abi;
  state[3] = Abo;
  state[4] = Abu;
  state[5] = Aga;
  state[6] = Age;
  state[7] = Agi;
  state[8] = Ago;
  state[9] = Agu;
  state[10] = Aka;
  state[11] = Ake;
  state[12] = Aki;
  state[13] = Ako;
  state[14] = Aku;
  state[15] = Ama;
  state[16] = Ame;
  state[17] = Ami;
  state[18] = Amo;
  state[19] = Amu;
  state[20] = Asa;
  state[21] = Ase;
  state[22] = Asi;
  state[23] = Aso;
  state[24] = Asu;

#undef round
}

#include <string.h>
#define MIN(a, b) ((a) < (b) ? (a) : (b))

__device__ static void keccak_absorb(uint64_t *s,
                          unsigned int r,
                          const unsigned char *m, unsigned long long int mlen,
                          unsigned char p)
{
  unsigned long long i;
  unsigned char t[200];

  while (mlen >= r)
  {
    for (i = 0; i < r / 8; ++i)
      s[i] ^= load64(m + 8 * i);

    KeccakF1600_StatePermute(s);
    mlen -= r;
    m += r;
  }

  for (i = 0; i < r; ++i)
    t[i] = 0;
  for (i = 0; i < mlen; ++i)
    t[i] = m[i];
  t[i] = p;
  t[r - 1] |= 128;
  for (i = 0; i < r / 8; ++i)
    s[i] ^= load64(t + 8 * i);
}

__device__ static void keccak_squeezeblocks(unsigned char *h, unsigned long long int nblocks,
                                 uint64_t *s,
                                 unsigned int r)
{
  unsigned int i;
  while (nblocks > 0)
  {
    KeccakF1600_StatePermute(s);
    for (i = 0; i < (r >> 3); i++)
    {
      store64(h + 8 * i, s[i]);
    }
    h += r;
    nblocks--;
  }
}
__device__ void shake128(unsigned char *output, unsigned long long outlen,
              const unsigned char *input, unsigned long long inlen)
{
  uint64_t s[25];
  unsigned char t[SHAKE128_RATE];
  unsigned long long nblocks = outlen / SHAKE128_RATE;
  size_t i;

  for (i = 0; i < 25; ++i)
    s[i] = 0;

  /* Absorb input */
  keccak_absorb(s, SHAKE128_RATE, input, inlen, 0x1F);

  /* Squeeze output */
  keccak_squeezeblocks(output, nblocks, s, SHAKE128_RATE);

  output += nblocks * SHAKE128_RATE;
  outlen -= nblocks * SHAKE128_RATE;

  if (outlen)
  {
    keccak_squeezeblocks(t, 1, s, SHAKE128_RATE);
    for (i = 0; i < outlen; i++)
      output[i] = t[i];
  }
}

__device__  void BS2POLq(const uint8_t bytes[SABER_POLYBYTES], uint16_t *data)
{
   size_t j, offset_byte, offset_data;
   for (j = 0; j < SABER_N / 8; j++)
   {
      offset_byte = 13 * j;
      offset_data = 8 * j;
      data[offset_data + 0] = (bytes[offset_byte + 0] & (0xff)) | ((bytes[offset_byte + 1] & 0x1f) << 8);
      data[offset_data + 1] = (bytes[offset_byte + 1] >> 5 & (0x07)) | ((bytes[offset_byte + 2] & 0xff) << 3) | ((bytes[offset_byte + 3] & 0x03) << 11);
      data[offset_data + 2] = (bytes[offset_byte + 3] >> 2 & (0x3f)) | ((bytes[offset_byte + 4] & 0x7f) << 6);
      data[offset_data + 3] = (bytes[offset_byte + 4] >> 7 & (0x01)) | ((bytes[offset_byte + 5] & 0xff) << 1) | ((bytes[offset_byte + 6] & 0x0f) << 9);
      data[offset_data + 4] = (bytes[offset_byte + 6] >> 4 & (0x0f)) | ((bytes[offset_byte + 7] & 0xff) << 4) | ((bytes[offset_byte + 8] & 0x01) << 12);
      data[offset_data + 5] = (bytes[offset_byte + 8] >> 1 & (0x7f)) | ((bytes[offset_byte + 9] & 0x3f) << 7);
      data[offset_data + 6] = (bytes[offset_byte + 9] >> 6 & (0x03)) | ((bytes[offset_byte + 10] & 0xff) << 2) | ((bytes[offset_byte + 11] & 0x07) << 10);
      data[offset_data + 7] = (bytes[offset_byte + 11] >> 3 & (0x1f)) | ((bytes[offset_byte + 12] & 0xff) << 5);
   }
}


__device__ void BS2POLVECq(const uint8_t bytes[SABER_POLYVECBYTES], uint16_t *data)
{
   size_t i;
   for (i = 0; i < SABER_L; i++)
   {
      BS2POLq(bytes + i * SABER_POLYBYTES, data + i*SABER_N);
   }
}

__device__ uint64_t load_littleendian(const uint8_t *x, int bytes)
{
  int i;
  uint64_t r = x[0];
  for (i = 1; i < bytes; i++)
    r |= (uint64_t)x[i] << (8 * i);
  return r;
}

__device__ void cbd_gpu(uint16_t s[SABER_N], const uint8_t buf[SABER_POLYCOINBYTES])
{
#if SABER_MU == 6
  uint32_t t, d, a[4], b[4];
  int i, j;

  for (i = 0; i < SABER_N / 4; i++)
  {
    t = load_littleendian(buf + 3 * i, 3);
    d = 0;
    for (j = 0; j < 3; j++)
      d += (t >> j) & 0x249249;

    a[0] = d & 0x7;
    b[0] = (d >> 3) & 0x7;
    a[1] = (d >> 6) & 0x7;
    b[1] = (d >> 9) & 0x7;
    a[2] = (d >> 12) & 0x7;
    b[2] = (d >> 15) & 0x7;
    a[3] = (d >> 18) & 0x7;
    b[3] = (d >> 21);

    s[4 * i + 0] = (uint16_t)(a[0] - b[0]);
    s[4 * i + 1] = (uint16_t)(a[1] - b[1]);
    s[4 * i + 2] = (uint16_t)(a[2] - b[2]);
    s[4 * i + 3] = (uint16_t)(a[3] - b[3]);
  }
#elif SABER_MU == 8 
  uint32_t t, d, a[4], b[4];
  int i, j;

  for (i = 0; i < SABER_N / 4; i++)
  {
    t = load_littleendian(buf + 4 * i, 4);
    d = 0;
    for (j = 0; j < 4; j++)
      d += (t >> j) & 0x11111111;

    a[0] = d & 0xf;
    b[0] = (d >> 4) & 0xf;
    a[1] = (d >> 8) & 0xf;
    b[1] = (d >> 12) & 0xf;
    a[2] = (d >> 16) & 0xf;
    b[2] = (d >> 20) & 0xf;
    a[3] = (d >> 24) & 0xf;
    b[3] = (d >> 28);

    s[4 * i + 0] = (uint16_t)(a[0] - b[0]);
    s[4 * i + 1] = (uint16_t)(a[1] - b[1]);
    s[4 * i + 2] = (uint16_t)(a[2] - b[2]);
    s[4 * i + 3] = (uint16_t)(a[3] - b[3]);
  }
#elif SABER_MU == 10
  uint64_t t, d, a[4], b[4];
  int i, j;

  for (i = 0; i < SABER_N / 4; i++)
  {
    t = load_littleendian(buf + 5 * i, 5);
    d = 0;
    for (j = 0; j < 5; j++)
      d += (t >> j) & 0x0842108421UL;

    a[0] = d & 0x1f;
    b[0] = (d >> 5) & 0x1f;
    a[1] = (d >> 10) & 0x1f;
    b[1] = (d >> 15) & 0x1f;
    a[2] = (d >> 20) & 0x1f;
    b[2] = (d >> 25) & 0x1f;
    a[3] = (d >> 30) & 0x1f;
    b[3] = (d >> 35);

    s[4 * i + 0] = (uint16_t)(a[0] - b[0]);
    s[4 * i + 1] = (uint16_t)(a[1] - b[1]);
    s[4 * i + 2] = (uint16_t)(a[2] - b[2]);
    s[4 * i + 3] = (uint16_t)(a[3] - b[3]);
  }
#else
#error "Unsupported SABER parameter."
#endif
}

__global__ void shake128_gpu(uint8_t *out, const uint8_t *in, size_t inlen, uint32_t outlen, uint32_t out_stride) 
{
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint8_t p = 0x1F;   // For absorb
    uint32_t r = 168;   // For shake128
    int const s = threadIdx.x % 5;
    __shared__ uint64_t A[25];
    __shared__ uint64_t C[25];
    __shared__ uint64_t D[25];
    __shared__ uint64_t d_data[25];
    uint32_t i, count=0;
    __shared__ uint8_t t[200];
    uint32_t nblocks = outlen / SHAKE128_RATE;
    outlen -= nblocks * SHAKE128_RATE;  // Remain one block?

    // Initialize arrays to zeros
    for (i = 0; i < 8; ++i) {
        if (tid < 25) 
        {
            t[i*25 + tid] = 0;
        }
    }
    if (tid < 25) 
    {
        A[tid] = 0; C[tid] = 0; D[tid] = 0; d_data[tid] = 0; 
    }

    // Absorb phase
    while (inlen >= r) 
    {         
        if(tid<17) d_data[tid] ^= load64(in + bid*SABER_INDCPA_PUBLICKEYBYTES + 8 * tid + count*r); //136 / 8
        if (tid < 25) {
              
            A[tid] = d_data[tid];
            for (int i = 0; i<NROUNDS; ++i) {
                C[tid] = A[s] ^ A[s + 5] ^ A[s + 10] ^ A[s + 15] ^ A[s + 20];
                D[tid] = C[b[20 + s]] ^ R64(C[b[5 + s]], 1, 63);
                C[tid] = R64(A[a[tid]] ^ D[b[tid]], ro[tid][0], ro[tid][1]);
                A[d[tid]] = C[c[tid][0]] ^ ((~C[c[tid][1]]) & C[c[tid][2]]);
                
                A[tid] ^= rc[(tid == 0) ? 0 : 1][i];
            }          
            d_data[tid] = A[tid];
        }        
        inlen -= r;
        count++;
    }

    if(tid==0) 
    {
        t[inlen] = p;
        t[r - 1] |= 128;
    }
    __syncthreads();
    // printf("%u %x\n", tid, d_data[tid]);
    uint32_t repeat = (inlen/blockDim.x)+1;
    if(repeat==0) repeat = 1;
    for (i = 0; i < repeat; i++)
    {
        if(tid < inlen) t[i*blockDim.x + tid] = in[i*blockDim.x + tid+ count*r];
        inlen-=blockDim.x;
        __syncthreads();
    }   
    
    if(tid < 21) d_data[tid] ^= load64(t + 8*tid);    
    // if(bid==0) if(tid < 25) printf("%u %lx %u\n", tid, d_data[tid], count);
    // if(tid < 25) printf("%u %d %u\n", tid, in[tid], inlen);
   // if(threadIdx.x==0) 
   // {
   //    printf("\n seed\n"); for (int i = 0; i < SABER_SEEDBYTES; i++) printf("%u ", in[i]);  
   // }

    // Squeeze phase
    if (tid < 25) 
    {
        A[tid] = 0; C[tid] = 0; D[tid] = 0; 
    }    
    count=0;
    while (nblocks > 0) {
        if (tid < 25) {              
            A[tid] = d_data[tid];
            for (int i = 0; i<NROUNDS; ++i) {
                C[tid] = A[s] ^ A[s + 5] ^ A[s + 10] ^ A[s + 15] ^ A[s + 20];
                D[tid] = C[b[20 + s]] ^ R64(C[b[5 + s]], 1, 63);
                C[tid] = R64(A[a[tid]] ^ D[b[tid]], ro[tid][0], ro[tid][1]);
                A[d[tid]] = C[c[tid][0]] ^ ((~C[c[tid][1]]) & C[c[tid][2]]);
                
                A[tid] ^= rc[(tid == 0) ? 0 : 1][i];
            }          
            d_data[tid] = A[tid];
            store64(out + bid*out_stride+ count*r + 8*tid, d_data[ tid]);       
        }                
        count++;
        nblocks--;
    }

    if (outlen) {
        if (tid < 25) {              
            A[tid] = d_data[tid];
            for (int i = 0; i<NROUNDS; ++i) {
                C[tid] = A[s] ^ A[s + 5] ^ A[s + 10] ^ A[s + 15] ^ A[s + 20];
                D[tid] = C[b[20 + s]] ^ R64(C[b[5 + s]], 1, 63);
                C[tid] = R64(A[a[tid]] ^ D[b[tid]], ro[tid][0], ro[tid][1]);
                A[d[tid]] = C[c[tid][0]] ^ ((~C[c[tid][1]]) & C[c[tid][2]]);
                
                A[tid] ^= rc[(tid == 0) ? 0 : 1][i];
            }          
            d_data[tid] = A[tid];         
        }             
        
        if(tid<outlen/8) store64(out + bid*out_stride+ count*r + 8*tid, d_data[tid]);
    }
}
// void GenMatrix(uint16_t A[SABER_L][SABER_L][SABER_N], const uint8_t seed[SABER_SEEDBYTES])
__global__ void GenMatrix_gpu(uint16_t *A, uint8_t *seed)
{
   uint8_t buf[SABER_L * SABER_POLYVECBYTES];
   int i;
   uint32_t tid = threadIdx.x;
   // printf("GenMatrix: %u\n", SABER_L * SABER_POLYVECBYTES);
   shake128(buf, sizeof(buf), seed + tid*SABER_SEEDBYTES, SABER_SEEDBYTES);

   for (i = 0; i < SABER_L; i++)
   {
      BS2POLVECq(buf + i * SABER_POLYVECBYTES, A + tid*SABER_L*SABER_L*SABER_N + i*SABER_L*SABER_N);
   }
}

__global__ void GenMatrix_gpu2(uint16_t *A, uint8_t *seed)
{
   uint8_t buf[SABER_L * SABER_POLYVECBYTES];
   int i;
   uint32_t tid = threadIdx.x;
   // printf("GenMatrix: %u\n", SABER_L * SABER_POLYVECBYTES);
   shake128(buf, sizeof(buf), seed + tid*SABER_INDCPA_PUBLICKEYBYTES, SABER_SEEDBYTES);

   for (i = 0; i < SABER_L; i++)
   {
      BS2POLVECq(buf + i * SABER_POLYVECBYTES, A + tid*SABER_L*SABER_L*SABER_N + i*SABER_L*SABER_N);
   }
}

// void GenSecret_gpu(uint16_t s[SABER_L][SABER_N], const uint8_t seed[SABER_NOISE_SEEDBYTES])
__global__ void GenSecret_gpu(uint16_t *s, uint8_t *seed)
{
   uint8_t buf[SABER_L * SABER_POLYCOINBYTES] = {0};
   size_t i;
   uint32_t tid = threadIdx.x;
   shake128(buf, sizeof(buf), seed + tid*64, SABER_NOISE_SEEDBYTES);

   for (i = 0; i < SABER_L; i++)
   {
      cbd_gpu(s + tid*SABER_N*SABER_L + i*SABER_N, buf + i * SABER_POLYCOINBYTES);
   }
}

// void sha3_256(unsigned char *output, const unsigned char *input, unsigned long long inlen)
__global__ void sha3_256_gpu(uint8_t *output, uint8_t *input, unsigned long long inlen, uint32_t in_stride, uint32_t out_stride)
{
  uint64_t s[25];
  unsigned char t[SHA3_256_RATE];
  size_t i;
   uint32_t tid = threadIdx.x;

  for (i = 0; i < 25; ++i)
    s[i] = 0;

  /* Absorb input */
  keccak_absorb(s, SHA3_256_RATE, input + tid*in_stride, inlen, 0x06);

  /* Squeeze output */
  keccak_squeezeblocks(t, 1, s, SHA3_256_RATE);

  for (i = 0; i < 32; i++)
    output[i + tid*out_stride] = t[i];
}

// void sha3_512(unsigned char *output, const unsigned char *input, unsigned long long inlen)
__global__ void sha3_512_gpu(uint8_t *output, uint8_t *input, unsigned long long inlen)
{
  uint64_t s[25];
  unsigned char t[SHA3_512_RATE];
  size_t i;
   uint32_t tid = threadIdx.x;
  for (i = 0; i < 25; ++i)
    s[i] = 0;

  /* Absorb input */
  keccak_absorb(s, SHA3_512_RATE, input + tid*64, inlen, 0x06);

  /* Squeeze output */
  keccak_squeezeblocks(t, 1, s, SHA3_512_RATE);

  for (i = 0; i < 64; i++)
    output[i + tid*64] = t[i];
}
